#include "hip/hip_runtime.h"
 #include <iostream>
 #include <iomanip>
 #include <cstdlib>
 #include <chrono>
 #include <cstdlib>
 #include <algorithm>
 #include <cmath>
 #include <random>
 // imports do thrust
  #include <thrust/host_vector.h>
 #include <thrust/device_vector.h>
 #include <thrust/generate.h>
 #include <thrust/functional.h>
 #include <thrust/copy.h> 

 using namespace std::chrono;

 void reportTime(const char* msg, steady_clock::duration span) {
     auto ms = duration_cast<milliseconds>(span);
     std::cout << msg << " - levou - " <<
      ms.count() << " milisegundos" << std::endl;
 }

// CRIAR UM FUNCTOR (TIPO UM KERNEL PARA CALCULAR O QUADRADO)

struct square {
    // não tem parametro interno, então não precisa nem de construtor de copia
    __host__ __device__
    float operator()(const float &x) const { // não é o x vetor, é a coordenada do vetor, só um ponto
        return x * x;
    }
};

 // IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
 float magnitude(thrust::device_vector<float> x) {
     thrust::transform(x.begin() , x.end() , x.begin() , square());  // se não colocar nada ou só o begin ele sobrescreve o resultado no próprio vetor   

     float soma_quadrados = thrust::reduce(x.begin(), x.end());
     float result = std::sqrt(soma_quadrados);
     return result;

     // outra solucao:
    //  return std::sqrt(thrust::transform_reduce(x.begin(), x.end(), square(), 0.0f, thrust::plus<float>()));
 }

 int main(int argc, char** argv) {
     if (argc != 2) {
         std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
         std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
         return 1;
     }
     int n = std::atoi(argv[1]); //numero de elementos
     steady_clock::time_point ts, te;

     // Faça um  vector em thrust 
     
     thrust::host_vector<float> v(n);

     // inicialize o  vector

     ts = steady_clock::now();

     thrust::generate(v.begin(), v.end(), rand);

     thrust::device_vector<float> v_d = v;

     te = steady_clock::now();
     reportTime("Inicializacao", te - ts);

     // Calcule a magnitude do vetor
     ts = steady_clock::now();
     float len = magnitude(v_d);
     te = steady_clock::now();
     reportTime("Tempo para calculo", te - ts);


     std::cout << std::fixed << std::setprecision(4);
     std::cout << "Magnitude : " << len << std::endl;
 }